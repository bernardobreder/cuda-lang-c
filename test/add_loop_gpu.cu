
#define N 10

int main( void ) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    cul_cudaMalloc( (void**)&dev_a, N * sizeof(int) );
    cul_cudaMalloc( (void**)&dev_b, N * sizeof(int) );
    cul_cudaMalloc( (void**)&dev_c, N * sizeof(int) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    cul_cudaMemory( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
    cul_cudaMemory( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );

    cul_recordExecution(add, 1, N, 1);
	add<<<N,1>>>( dev_a, dev_b, dev_c );

    // copy the array 'c' back from the GPU to the CPU
    cul_cudaMemory( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost );

    // display the results
    for (int i=0; i<N; i++) {
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }

    // free the memory allocated on the GPU
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );

    return 0;
}
